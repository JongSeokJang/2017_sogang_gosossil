#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <Windows.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START() { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START()
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
	CUDA_CALL(hipEventCreate(&cuda_timer_start));
	CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
	CUDA_CALL( hipEventDestroy( cuda_timer_start ) );
	CUDA_CALL( hipEventDestroy( cuda_timer_stop ) );
}

inline void start_device_timer()
{
	hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
	TIMER_T ms;
	hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
	hipEventSynchronize(cuda_timer_stop);

	hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
	return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif


#define BLOCK_SIZE 32
#define DELTA 0.000001
int n;
TIMER_T compute_time = 0;
TIMER_T device_time = 0;

__host__ void cuda_error_check(const char * prefix, const char * postfix)
{
	if (hipPeekAtLastError() != hipSuccess)
	{
		printf("%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		//wait_exit();
		exit(1);
	}
}

void find_roots_CPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1, int n) {

	int i;
	float a, b, c, d, x0, x1, tmp;

	for(i = 0; i < n; i++) {
		a = A[i]; 
		b = B[i]; 
		c = C[i];

		d = sqrtf(b*b - 4.0f*a*c);
		tmp = 1.0f / (2.0f*a);
		x0 = (-b - d) * tmp;
		x1 = (-b + d) * tmp;

		if(x0 <= x1) {

			X0[i] = x0; 
			X1[i] = x1;
			FX0[i] = (a*x0 + b)*x0 + c;
			FX1[i] = (a*x1 + b)*x1 + c;

		} else {
			X0[i] = x1; 
			X1[i] = x0;
			FX0[i] = (a*x1 + b)*x1 + c;
			FX1[i] = (a*x0 + b)*x0 + c;

		}
	}
}


__global__ void find_roots_Kernel(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1)
{
	
	unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

	unsigned int tid = gridDim.x * blockDim.x * row + col;
	
	float a, b, c, d, x0, x1, tmp;

	a = A[tid]; 
	b = B[tid]; 
	c = C[tid];

	d = sqrtf(b*b - 4.0f*a*c);
	tmp = 1.0f / (2.0f*a);
	x0 = (-b - d) * tmp;
	x1 = (-b + d) * tmp;
	if(x0 <= x1) {
		X0[tid] = x0; X1[tid] = x1;
		FX0[tid] = (a*x0 + b)*x0 + c;
		FX1[tid] = (a*x1 + b)*x1 + c;
	} else {
		X0[tid] = x1; X1[tid] = x0;
		FX0[tid] = (a*x1 + b)*x1 + c;
		FX1[tid] = (a*x0 + b)*x0 + c;
	}

}


void find_roots_GPU(float *A, float *B, float *C, float *X0, float *X1, float *FX0, float *FX1) 
{
	int n1, n2;
	float *d_A, *d_B, *d_C, *d_X0, *d_X1, *d_FX0, *d_FX1;
	size_t size;

	size = n*sizeof(float);

	CUDA_CALL( hipMalloc(&d_A, size) );
	CUDA_CALL( hipMalloc(&d_B, size) );
	CUDA_CALL( hipMalloc(&d_C, size) );
	CUDA_CALL( hipMalloc(&d_X0, size) );
	CUDA_CALL( hipMalloc(&d_X1, size) );
	CUDA_CALL( hipMalloc(&d_FX0, size) );
	CUDA_CALL( hipMalloc(&d_FX1, size) );

	CUDA_CALL( hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
	CUDA_CALL( hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
	CUDA_CALL( hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

	n1 = 1024;
	n2 = n/1024;

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid( n1/dimBlock.x, n2/dimBlock.y);
	
	CHECK_TIME_INIT_GPU()
	CHECK_TIME_START_GPU()
	find_roots_Kernel <<< dimGrid, dimBlock >>> (d_A, d_B, d_C, d_X0, d_X1, d_FX0, d_FX1);
	CHECK_TIME_END_GPU(device_time);
	CHECK_TIME_DEST_GPU()

	CUDA_CALL( hipMemcpy( X0, d_X0, size, hipMemcpyDeviceToHost))
	CUDA_CALL( hipMemcpy( X1, d_X1, size, hipMemcpyDeviceToHost))
	CUDA_CALL( hipMemcpy( FX0, d_FX0, size, hipMemcpyDeviceToHost))
	CUDA_CALL( hipMemcpy( FX1, d_FX1, size, hipMemcpyDeviceToHost))

}

void main(void) {

	FILE *fp_A = fopen("A.bin", "rb");
	FILE *fp_B = fopen("B.bin", "rb");
	FILE *fp_C = fopen("C.bin", "rb");

	if(fp_A == NULL || fp_B == NULL || fp_C == NULL ) {
		printf(" - Cannot open abc.bin\n\n");
		return;
	}

	fread(&n, sizeof(int), 1, fp_A);
	fread(&n, sizeof(int), 1, fp_B);
	fread(&n, sizeof(int), 1, fp_C);

	float* A = new float[n];
	float* B = new float[n];
	float* C = new float[n];

	fread(A, sizeof(float), n, fp_A);
	fread(B, sizeof(float), n, fp_B);
	fread(C, sizeof(float), n, fp_C);

	float *X0 = new float[n];
	float *X1 = new float[n];
	float *FX0 = new float[n];
	float *FX1 = new float[n];

	float *X0_G = new float[n];
	float *X1_G = new float[n];
	float *FX0_G = new float[n];
	float *FX1_G = new float[n];

	CHECK_TIME_START();
	find_roots_CPU(A, B, C, X0, X1, FX0, FX1, n);
	CHECK_TIME_END( compute_time );
	printf("***CPU [%lf] Time taken = %.6fms\n",X0[n-1], compute_time);
	CUDA_CALL(hipSetDevice(0));
	find_roots_GPU(A, B, C, X0_G, X1_G, FX0_G, FX1_G);
	printf("***GPU [%lf] Time taken = %.6fms\n",X0_G[n-1], device_time);

	FILE* fp_x0 = fopen( "X0.bin", "wb" );
	FILE* fp_x1 = fopen( "X1.bin", "wb" );
	FILE* fp_fx0 = fopen( "FX0.bin", "wb" );
	FILE* fp_fx1 = fopen( "FX1.bin", "wb" );


	fwrite(X0_G, sizeof(float), n, fp_x0);
	fwrite(X1_G, sizeof(float), n, fp_x1);
	fwrite(FX0_G, sizeof(float), n, fp_fx0);
	fwrite(FX1_G, sizeof(float), n, fp_fx1);

	fclose( fp_A );
	fclose( fp_B );
	fclose( fp_C );

	fclose(fp_x0);
	fclose(fp_x1);
	fclose(fp_fx0);
	fclose(fp_fx1);

	delete[] A;
	delete[] B;
	delete[] C;
	delete[] X0;
	delete[] X1;
	delete[] FX0;
	delete[] FX1;

	delete[] X0_G;
	delete[] X1_G;
	delete[] FX0_G;
	delete[] FX1_G;

}
